#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

template <typename T> T ceil_div(const T x, const T y) { return x / y + !!(x % y); }

__global__ void gradient_stabilized_cuda_forward_kernel(
    const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> input,    // (batch_size, input_length)
    const torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> mapping,    // (num_luts, n)
    const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> luts,     // (num_luts, 2^n)
    torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> output) {       // (batch_size, num_luts)
    
    const int batch_size = output.size(0);
    const int num_luts = output.size(1);

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < batch_size; i += blockDim.x * gridDim.x) {
        for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < num_luts; j += blockDim.y * gridDim.y) {
                
            // Threshold at 0.5: input >= 0.5 -> 1, input < 0.5 -> 0
            uint addr = input[i][mapping[j][0]] >= 0.5f;
            for(int l = 1; l < mapping.size(1); ++l)
                addr |= (uint)(input[i][mapping[j][l]] >= 0.5f) << l;

            output[i][j] = luts[j][addr];
    
        };
    };

}

torch::Tensor dwn_stable_cuda_forward(
    torch::Tensor input_tensor,
    torch::Tensor mapping_tensor,
    torch::Tensor luts_tensor) {
  
    auto batch_size = input_tensor.size(0);
    auto output_size = luts_tensor.size(0);

    auto output_tensor = torch::empty({batch_size, output_size}, 
        torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, input_tensor.device().index()));

    dim3 threads_per_block(32, 32);

    dim3 blocks_per_grid(
        min(static_cast<int64_t>(65535), ceil_div(batch_size, static_cast<int64_t>(threads_per_block.x))),
        min(static_cast<int64_t>(65535), ceil_div(output_size, static_cast<int64_t>(threads_per_block.y)))
    );

    gradient_stabilized_cuda_forward_kernel<<<blocks_per_grid, threads_per_block>>>(
        input_tensor.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
        mapping_tensor.packed_accessor32<int, 2, torch::RestrictPtrTraits>(),
        luts_tensor.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
        output_tensor.packed_accessor32<float, 2, torch::RestrictPtrTraits>()
    );

    hipDeviceSynchronize();

    return output_tensor;
}

__global__ void gradient_stabilized_cuda_backward_kernel(
    const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> input,          // (batch_size, input_length)
    const torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> mapping,          // (num_luts, n)
    const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> luts,           // (num_luts, 2^n)
    const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> output_grad,    // (batch_size, num_luts)
    const float gradient_scale,                                                              // scalar scaling factor
    torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> input_grad,           // (batch_size, input_length) 
    torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> luts_grad) {          // (num_luts, 2^n)
          

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < output_grad.size(0); i += blockDim.x * gridDim.x) {
        for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < output_grad.size(1); j += blockDim.y * gridDim.y) {

            // LUT grad - threshold at 0.5
            uint addr = input[i][mapping[j][0]] >= 0.5f;
            for(int l = 1; l < mapping.size(1); ++l) {
                addr |= (uint)(input[i][mapping[j][l]] >= 0.5f) << l;
            };
            
            // Apply gradient scaling to LUT gradient
            atomicAdd(&luts_grad[j][addr], output_grad[i][j] * gradient_scale);

            // Input grad using Extended Finite Difference (EFD) with scaling
            // Iterate over ALL 2^n possible addresses with Hamming distance weighting
            const int n = mapping.size(1);
            const int lut_size = 1 << n;
            
            for(int l = 0; l < n; ++l) {
                float total_gradient = 0.0f;
                
                // Create mask to exclude l-th bit for Hamming distance calculation
                uint mask = ((1 << n) - 1) & ~(1 << l);
                uint addr_masked = addr & mask;
                
                // Iterate over all possible addresses k
                for(uint k = 0; k < lut_size; ++k) {
                    // Calculate Hamming distance between addr and k, excluding l-th bit
                    uint k_masked = k & mask;
                    int hamming_dist = __popc(addr_masked ^ k_masked);
                    
                    // Get k_l (l-th bit of k)
                    uint k_l = (k >> l) & 1;
                    
                    // Calculate sign factor: (-1)^(1-k_l)
                    float sign_factor = (k_l == 0) ? -1.0f : 1.0f;
                    
                    // Get LUT value at position k
                    float lut_value = luts[j][k];
                    
                    // Add weighted contribution
                    total_gradient += sign_factor * lut_value / (hamming_dist + 1.0f);
                }
                
                // Apply gradient scaling to input gradient
                atomicAdd(&input_grad[i][mapping[j][l]], total_gradient * output_grad[i][j] * gradient_scale);
            };

        };
    };

};

std::vector<torch::Tensor> dwn_stable_cuda_backward(
    torch::Tensor input_tensor,
    torch::Tensor mapping_tensor,
    torch::Tensor luts_tensor,
    torch::Tensor output_grad_tensor,
    float gradient_scale) {
  
    auto batch_size = output_grad_tensor.size(0);
    auto output_size = output_grad_tensor.size(1);

    auto input_grad_tensor = torch::zeros_like(input_tensor);
    auto luts_grad_tensor = torch::zeros_like(luts_tensor);

    dim3 threads_per_block(32, 32);

    dim3 blocks_per_grid(
        min(static_cast<int64_t>(65535), ceil_div(batch_size, static_cast<int64_t>(threads_per_block.x))),
        min(static_cast<int64_t>(65535), ceil_div(output_size, static_cast<int64_t>(threads_per_block.y)))
    );

    gradient_stabilized_cuda_backward_kernel<<<blocks_per_grid, threads_per_block>>>(
        input_tensor.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
        mapping_tensor.packed_accessor32<int, 2, torch::RestrictPtrTraits>(),
        luts_tensor.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
        output_grad_tensor.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
        gradient_scale,
        input_grad_tensor.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
        luts_grad_tensor.packed_accessor32<float, 2, torch::RestrictPtrTraits>()
    );

    hipDeviceSynchronize();

    return {input_grad_tensor, luts_grad_tensor};
}
